#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "src/frog.cuh"
#include "src/vwarp.cu"

#define SOURCE_VERTEX 3

// print info about bfs values
void print_bfs_values(const int * const values, int const size) {
	int visited = 0;
	int step = 0;
	int first = 0;
	// get the max step and count the visited
	for (int i = 0; i < size; i++) {
		if (values[i] != 0) {
			visited++;
			if (values[i] > step) step = values[i];
			if (values[i] == 1) first = i;
		}
	}
	// count vertices of each step
	if (step == 0) return;
	int * m = (int *) calloc(step + 1, sizeof(int));
	for (int i = 0; i < size; i++) {
		m[values[i]]++;
	}
	// print result info
	printf("\tSource = %d, Step = %d, Visited = %d\n", first, step, visited);
	printf("\tstep\tvisit\n");
	for (int i = 1; i <= step; i++) {
		printf("\t%d\t%d\n", i, m[i]);
	}
	free(m);
}

// check if arrays v1 & v2 have the same first n elements (no boundary check)
static void check_values(const int * const v1, const int * const v2, int n) {
	for (int i = 0; i < n; i++) {
		if (v1[i] != v2[i]) {
			printf("Check Fail\n");
			return;
		}
	}
	printf("Check PASS\n");
}

static void cpu_bfs_vertex(
		const int vertex_num,
		const int * const vertex_begin,
		const int * const edge_dest,
		int * const values,
		const int first_vertex
		) {

	timer_start();
	// for simplicity, use a large but simple queue instead of a small full functional queue)
	int * queue = (int *)calloc(vertex_num, sizeof(int));
	if (queue == NULL) {
		perror("Out of memory");
		exit(1);
	}
	// the position to put next enqueue element & get next dequeue element
	int incount = 0;
	int outcount = 0;
	// initialization
	memset(values, 0, vertex_num * sizeof(int));
	values[first_vertex] = 1;
	queue[incount++] = first_vertex;

	int step = 0;
	while (incount > outcount) {
		// dequeue the vertex to be visited
		int v = queue[outcount++];
		step = values[v];
		for (int e = vertex_begin[v]; e < vertex_begin[v + 1]; e++) {
			int dest = edge_dest[e];
			if (values[dest] == 0) {
				// enqueue the vertex will be visited
				values[dest] = step + 1;
				queue[incount++] = dest;
			}
		}
	}
	printf("\t%.2f\tcpu_bfs_vertex\tStep=%d\n", timer_stop(), step);
	free(queue);
}

static void cpu_bfs_edge(
		const int edge_num,
		const int vertex_num,
		const int * const edge_src,
		const int * const edge_dest,
		int * const values,
		const int first_vertex
		) {

	timer_start();
	// for simplicity, use a large but simple queue instead of a small full functional queue)
	int * queue = (int *)calloc(vertex_num, sizeof(int));
	if (queue == NULL) {
		perror("Out of memory");
		exit(1);
	}
	// the position to put next enqueue element & get next dequeue element
	int incount = 0;
	int outcount = 0;
	// initialization
	memset(values, 0, vertex_num * sizeof(int));
	values[first_vertex] = 1;
	queue[incount++] = first_vertex;

	int step = 0;
	int ite=0;
	while (incount > outcount) {
		// dequeue the vertex to be visited
		int v = queue[outcount++];
		step = values[v];
		for (int e=0; e < edge_num; e++) {
			int src = edge_src[e];
			int dest= edge_dest[e];
			if (src ==v && values[dest] == 0) {
				// enqueue the vertex will be visited
				values[dest] = step + 1;
				queue[incount++] = dest;
			}
		}
		ite++;
	}
	printf("\t%.2f\tcpu_bfs_edge\tStep=%d\n", timer_stop(), step);
	free(queue);
}

// BFS kernel run on edges with inner loop
static __global__ void kernel_edge_loop(
		const int edge_num,
		const int * const edge_src,
		const int * const edge_dest,
		int * const values,
		const int step,
		int * const continue_flag
		) {
	// total thread number & thread index of this thread
	int n = blockDim.x * gridDim.x;
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	// step counter
	int curStep = step;
	int nextStep = curStep + 1;
	// continue flag for each thread
	int flag = 0;
	// proceeding loop
	for (int i = index; i < edge_num; i += n) {
		if (values[edge_src[i]] == curStep && values[edge_dest[i]] == 0) {
			values[edge_dest[i]] = nextStep;
			flag = 1;
		}
	}
	// update flag
	if (flag == 1) *continue_flag = 1;
}

// BFS kernel run on edges without inner loop
static __global__ void kernel_edge(
		const int edge_num,
		const int * const edge_src,
		const int * const edge_dest,
		int * const values,
		const int step,
		int * const continue_flag
		) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	// step counter
	int curStep = step;
	int nextStep = curStep + 1;
	// proceeding loop
	if(index < edge_num) {
		if (values[edge_src[index]] == curStep && values[edge_dest[index]] == 0) {
			values[edge_dest[index]] = nextStep;
			*continue_flag=1;
		}
	}
}
// BFS algorithm on graph g, not partitioned, run on edges with inner loop
static void gpu_bfs_edge(
		const Graph * const g,
		int * const values,
		int const first_vertex
		) {
	int step = 1, flag = 0;
	float bfsTime = 0.0;
	timer_start();
	int vertex_num = g->vertex_num;
	int edge_num = g->edge_num;
	Auto_Utility();
	// Allocate GPU buffer
	CudaBufferCopy(int, dev_edge_src, edge_num, g->edge_src);
	CudaBufferCopy(int, dev_edge_dest, edge_num, g->edge_dest);
	CudaBufferZero(int, dev_value, vertex_num);
	CudaBufferZero(int, dev_continue_flag, 1)
		// Set Source Vertex Value (Little Endian)
		CudaMemset(dev_value + first_vertex, 1, 1);
	// Main Loop
	do {
		// Clear Flag
		CudaMemset(dev_continue_flag, 0, sizeof(int));
		// Launch Kernel for this Iteration
		CudaTimerBegin();
		kernel_edge<<<(edge_num + 255) / 256, 256>>>
			(
			 edge_num,
			 dev_edge_src,
			 dev_edge_dest,
			 dev_value,
			 step,
			 dev_continue_flag
			);
		bfsTime += CudaTimerEnd();
		// Copy Back Flag
		CudaMemcpyD2H(&flag, dev_continue_flag, sizeof(int));
		step++;
	} while(flag);
	// Copy Back Values
	CudaMemcpyD2H(values, dev_value, vertex_num * sizeof(int));
	printf("\t%.2f\t%.2f\tgpu_bfs_edge\tstep=%d\t", bfsTime, timer_stop(), step - 1);
}

// BFS algorithm on graph g, not partitioned, run on edges with inner loop
static void gpu_bfs_edge_loop(
		const Graph * const g,
		int * const values,
		int const first_vertex
		) {
	int step = 1, flag = 0;
	float bfsTime = 0.0;
	timer_start();
	int vertex_num = g->vertex_num;
	int edge_num = g->edge_num;
	Auto_Utility();
	// Allocate GPU buffer
	CudaBufferCopy(int, dev_edge_src, edge_num, g->edge_src);
	CudaBufferCopy(int, dev_edge_dest, edge_num, g->edge_dest);
	CudaBufferZero(int, dev_value, vertex_num);
	CudaBufferZero(int, dev_continue_flag, 1)
		// Set Source Vertex Value (Little Endian)
		CudaMemset(dev_value + first_vertex, 1, 1);
	// Main Loop
	do {
		// Clear Flag
		CudaMemset(dev_continue_flag, 0, sizeof(int));
		// Launch Kernel for this Iteration
		CudaTimerBegin();
		kernel_edge_loop<<<208, 128>>>
			(
			 edge_num,
			 dev_edge_src,
			 dev_edge_dest,
			 dev_value,
			 step,
			 dev_continue_flag
			);
		bfsTime += CudaTimerEnd();
		// Copy Back Flag
		CudaMemcpyD2H(&flag, dev_continue_flag, sizeof(int));
		step++;
	} while(flag);
	// Copy Back Values
	CudaMemcpyD2H(values, dev_value, vertex_num * sizeof(int));
	printf("\t%.2f\t%.2f\tgpu_bfs_edge_loop\tstep=%d\t", bfsTime, timer_stop(), step - 1);
}



// BFS kernel run on vertices with inner loop
static __global__ void kernel_vertex_loop(
		const int vertex_num,
		const int * const vertex_begin,
		const int * const edge_dest,
		int * const values,
		const int step,
		int * const continue_flag
		) {
	// total thread number & thread index of this thread
	int n = blockDim.x * gridDim.x;
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	// step counter
	int curStep = step;
	int nextStep = curStep + 1;
	// continue flag for each thread
	int flag = 0;
	// proceeding loop
	for (int i = index; i < vertex_num; i += n) {
		if (values[i] == curStep) {
			for (int k = vertex_begin[i]; k < vertex_begin[i + 1]; k++) {
				int dest = edge_dest[k];
				if (values[dest] == 0) {
					values[dest] = nextStep;
					flag = 1;
				}
			}
		}
	}
	if (flag) *continue_flag = 1;
}

// BFS kernel run on vertices without inner loop
static __global__ void kernel_vertex(
		const int vertex_num,
		const int * const vertex_begin,
		const int * const edge_dest,
		int * const values,
		const int step,
		int * const continue_flag
		) {
	// total thread number & thread index of this thread
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	// proceed
	if (i < vertex_num) {
		if (values[i] == step) {
			for (int k = vertex_begin[i]; k < vertex_begin[i + 1]; k++) {
				int dest = edge_dest[k];
				if (values[dest] == 0) {
					values[dest] = step + 1;
					*continue_flag = 1;
				}
			}
		}
	}
}

// virtual warp by myself 
static __global__ void kernel_vertex_vwarp (
		int const vertex_num,
		int const step,
		const int * const vertex_begin,
		const int * const edge_dest,
		int * const values,
		int * const continue_flag
		) {
#define VWARP_SIZE 2
	// thread index of this thread
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int n=blockDim.x*gridDim.x;
	int vwarp_num = n / VWARP_SIZE;
	int vwarp_id = index / VWARP_SIZE;
	int vwarp_tid = index % VWARP_SIZE;
	// offset of each vwarp for cache indexing (vwarp_offset + vwarp_tid == threadIdx.x)
	int vwarp_offset = threadIdx.x / VWARP_SIZE * VWARP_SIZE;
	// proceed
	for(int i=vwarp_id;i<vertex_num;i+=vwarp_num) {
		int id = i;
		int new_value = values[id];
		int flag = 0;
		if (new_value==step)
		{

			for (int e = vertex_begin[i]+vwarp_tid; e < vertex_begin[i + 1]; e+=VWARP_SIZE) {
				int dest_value = values[edge_dest[e]];
				if (dest_value==0)
				{           
					flag=1;
					values[edge_dest[e]]=step+1;
				}
			}
		}
		// update values
		if (flag) {
			*continue_flag = 1;
		}
	}
}

//virtual warp similar to Totem
template<int VWARP_WIDTH1,int VWARP_BATCH1>
static  __global__ void kernel_vertex_vwarp_totem (
		int const vertex_num,
		int const step,
		const int * const vertex_begin,
		const int * const edge_dest,
		int * const values,
		int * const continue_flag
		) {
	if(THREAD_GLOBAL_INDEX>=
			vwarp_thread_count(vertex_num,VWARP_WIDTH1,VWARP_BATCH1)){return ;}
	__shared__ bool finish_block;
	finish_block=true;
	__syncthreads();

	int start_vertex=vwarp_block_start_vertex(VWARP_WIDTH1,VWARP_BATCH1)+
		vwarp_warp_start_vertex(VWARP_WIDTH1,VWARP_BATCH1);
	int end_vertex=start_vertex+
		vwarp_warp_batch_size(vertex_num,VWARP_WIDTH1,VWARP_BATCH1);
	int warp_offset=vwarp_thread_index(VWARP_WIDTH1);
	// proceed
	for(int i=start_vertex;i<end_vertex;i++) {
		int id = i;
		int new_value = values[id];
		if(new_value==step)
		{
			const int nbr_count=vertex_begin[i+1]-vertex_begin[i];
			const int *edge=edge_dest+vertex_begin[i];
			for (int e =warp_offset; e < nbr_count; e+=VWARP_WIDTH1)
			{
				int nbr=edge[e];
				int dest_value = values[nbr];
				if (dest_value==0) 
				{
					values[nbr]=step+1;
					finish_block=false;
				}
			} 
		}

	}
	//__syncthreads();
	//if(!finish_block&&THREAD_GLOBAL_INDEX==0) *continue_flag=1;
	if(!finish_block) *continue_flag=1;
}


static void gpu_bfs_vertex_vwarp(
		const Graph * const g,
		int * const values,
		int const first_vertex
		){
	Auto_Utility();
	timer_start();
	int vertex_num = g->vertex_num;
	int edge_num = g->edge_num;
	// GPU buffer

	CudaBufferCopy(int, dev_vertex_begin, vertex_num, g->vertex_begin);
	CudaBufferCopy(int, dev_edge_dest, edge_num, g->edge_dest);
	CudaBufferZero(int, dev_value, vertex_num);
	CudaBufferZero(int, dev_continue_flag, 1);
	CudaMemset(dev_value + first_vertex, 1, 1);
	// settings
	int bn = (vertex_num + 255) / 256;
	int tn = 256;
	int flag = 0;
	int step = 1;
	float execTime = 0.0;

	const int threads=MAX_THREADS_PER_BLOCK;
	dim3 blocks;
	// Main Loop
	do {
		// Clear Flag
		CudaMemset(dev_continue_flag, 0, sizeof(int));
		kernel_configure(vwarp_thread_count(g->vertex_num,VWARP_WIDTH,BATCH_SIZE),
				blocks,threads);
		CudaTimerBegin();
		kernel_vertex_vwarp<<<blocks,threads>>>(
				vertex_num,
				step,
				dev_vertex_begin,
				dev_edge_dest,
				dev_value,
				dev_continue_flag
				);
		execTime += CudaTimerEnd();
		// Copy Back Flag
		CudaMemcpyD2H(&flag, dev_continue_flag, sizeof(int));
		step++;
	} while(flag != 0 && step < 1000);
	// Copy Back Values
	CudaMemcpyD2H(values, dev_value, vertex_num * sizeof(int));
	printf("\t%.2f\t%.2f\tgpu_bfs_vertex_vwarp\tStep=%d\t",
			execTime, timer_stop(), step);
}


static void gpu_bfs_vertex_vwarp_totem(
		const Graph * const g,
		int * const values,
		int const first_vertex
		){
	Auto_Utility();
	timer_start();
	int vertex_num = g->vertex_num;
	int edge_num = g->edge_num;
	// GPU buffer

	CudaBufferCopy(int, dev_vertex_begin, vertex_num, g->vertex_begin);
	CudaBufferCopy(int, dev_edge_dest, edge_num, g->edge_dest);
	CudaBufferZero(int, dev_value, vertex_num);
	CudaBufferZero(int, dev_continue_flag, 1);
	CudaMemset(dev_value + first_vertex, 1, 1);
	// settings
	int bn = (vertex_num + 255) / 256;
	int tn = 256;
	int flag = 0;
	int step = 1;
	float execTime = 0.0;

	const int threads=MAX_THREADS_PER_BLOCK;
	dim3 blocks;
	// Main Loop
	do {
		// Clear Flag
		CudaMemset(dev_continue_flag, 0, sizeof(int));
		kernel_configure(vwarp_thread_count(g->vertex_num,VWARP_WIDTH,BATCH_SIZE),
				blocks,threads);
		CudaTimerBegin();
		kernel_vertex_vwarp_totem<VWARP_WIDTH,BATCH_SIZE><<<blocks,threads>>>(
				vertex_num,
				step,
				dev_vertex_begin,
				dev_edge_dest,
				dev_value,
				dev_continue_flag
				);
		execTime += CudaTimerEnd();
		// Copy Back Flag
		CudaMemcpyD2H(&flag, dev_continue_flag, sizeof(int));
		step++;
	} while(flag != 0 && step < 1000);
	// Copy Back Values
	CudaMemcpyD2H(values, dev_value, vertex_num * sizeof(int));
	printf("\t%.2f\t%.2f\tgpu_bfs_vertex_vwarp_totem\tStep=%d\t",
			execTime, timer_stop(), step);
}

// BFS algorithm on graph g, not partitioned, run on vertices without inner loop
static void gpu_bfs_vertex(
		const Graph * const g,
		int * const values,
		int const first_vertex
		) {
	int step = 1, flag = 0;
	float bfsTime = 0.0;
	timer_start();
	int vertex_num = g->vertex_num;
	int edge_num = g->edge_num;
	Auto_Utility();
	// Allocate GPU buffer
	CudaBufferCopy(int, dev_vertex_begin, vertex_num + 1, g->vertex_begin);
	CudaBufferCopy(int, dev_edge_dest, edge_num, g->edge_dest);
	CudaBufferZero(int, dev_value, vertex_num);
	CudaBufferZero(int, dev_continue_flag, 1)
		// Set Source Vertex Value (Little Endian)
		CudaMemset(dev_value + first_vertex, 1, 1);
	// Main Loop
	do {
		// Clear Flag
		CudaMemset(dev_continue_flag, 0, sizeof(int));
		// Launch Kernel for this Iteration
		CudaTimerBegin();
		kernel_vertex<<<(vertex_num + 255) / 256, 256>>>
			(
			 vertex_num,
			 dev_vertex_begin,
			 dev_edge_dest,
			 dev_value,
			 step,
			 dev_continue_flag
			);
		bfsTime += CudaTimerEnd();
		// Copy Back Flag
		CudaMemcpyD2H(&flag, dev_continue_flag, sizeof(int));
		step++;
	} while(flag);
	// Copy Back Values
	CudaMemcpyD2H(values, dev_value, vertex_num * sizeof(int));
	printf("\t%.2f\t%.2f\tgpu_bfs_vertex\tstep=%d\t", bfsTime, timer_stop(), step - 1);
}

// BFS algorithm on graph g, not partitioned, run on vertices with inner loop
static void gpu_bfs_vertex_loop(
		const Graph * const g,
		int * const values,
		int const first_vertex
		) {
	int step = 1, flag = 0;
	float bfsTime = 0.0;
	timer_start();
	int vertex_num = g->vertex_num;
	int edge_num = g->edge_num;
	Auto_Utility();
	// Allocate GPU buffer
	CudaBufferCopy(int, dev_vertex_begin, vertex_num + 1, g->vertex_begin);
	CudaBufferCopy(int, dev_edge_dest, edge_num, g->edge_dest);
	CudaBufferZero(int, dev_value, vertex_num);
	CudaBufferZero(int, dev_continue_flag, 1)
		// Set Source Vertex Value (Little Endian)
		CudaMemset(dev_value + first_vertex, 1, 1);
	// Main Loop
	do {
		// Clear Flag
		CudaMemset(dev_continue_flag, 0, sizeof(int));
		// Launch Kernel for this Iteration
		CudaTimerBegin();
		kernel_vertex_loop<<<208, 256>>>
			(
			 vertex_num,
			 dev_vertex_begin,
			 dev_edge_dest,
			 dev_value,
			 step,
			 dev_continue_flag
			);
		bfsTime += CudaTimerEnd();
		// Copy Back Flag
		CudaMemcpyD2H(&flag, dev_continue_flag, sizeof(int));
		step++;
	} while(flag);
	// Copy Back Values
	CudaMemcpyD2H(values, dev_value, vertex_num * sizeof(int));
	printf("\t%.2f\t%.2f\tgpu_bfs_vertex_loop\tstep=%d\t", bfsTime, timer_stop(), step - 1);
}


// experiments of BFS on Graph g with Partition Table t and partitions
void bfs_experiments(const Graph * const g) {

	// partition on the Graph
	printf("-------------------------------------------------------------------\n");

	int * value_cpu = (int *) calloc(g->vertex_num, sizeof(int));
	int * value_gpu = (int *) calloc(g->vertex_num, sizeof(int));
	if (value_cpu == NULL || value_gpu == NULL) {
		perror("Out of Memory for values");
		exit(1);
	}

	printf("\tTime\tTotal\tTips\n");

	//	cpu_bfs_edge(g->edge_num,g->vertex_num, g->edge_src, g->edge_dest, value_cpu, SOURCE_VERTEX);
	cpu_bfs_vertex(g->vertex_num, g->vertex_begin, g->edge_dest, value_cpu, SOURCE_VERTEX);
	//  print_bfs_values(value_cpu, g->vertex_num);

	gpu_bfs_edge(g, value_gpu, SOURCE_VERTEX);
	check_values(value_cpu, value_gpu, g->vertex_num);
	gpu_bfs_edge_loop(g, value_gpu, SOURCE_VERTEX);
	check_values(value_cpu, value_gpu, g->vertex_num);

	gpu_bfs_vertex(g, value_gpu, SOURCE_VERTEX);
	check_values(value_cpu, value_gpu, g->vertex_num);
	gpu_bfs_vertex_vwarp(g, value_gpu, SOURCE_VERTEX);
	check_values(value_cpu, value_gpu, g->vertex_num);
	gpu_bfs_vertex_vwarp_totem(g, value_gpu, SOURCE_VERTEX);
	check_values(value_cpu, value_gpu, g->vertex_num);

	gpu_bfs_vertex_loop(g, value_gpu, SOURCE_VERTEX);
	check_values(value_cpu, value_gpu, g->vertex_num);

	free(value_cpu);
	free(value_gpu);
}


